
#include <hip/hip_runtime.h>
template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { return a + b; }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { return max(a, b); }
};

template <template <typename> class ReductionOp, typename T>
__inline__ __device__ T warpReduce(T val)
{
    for (int mask = 32 / 2; mask > 0; mask /= 2)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template <template <typename> class ReductionOp, typename T>
__inline__ __device__ T blockReduce(T val)
{
    int tid = threadIdx.x;
    int warp_id = tid / 32;
    int lane_id = tid % 32;
    int warp_nums = (blockDim.x + 31) / 32;
    static __shared__ T warp[64];
    val = warpReduce<ReductionOp, T>(val);
    if (lane_id == 0)
    {
        warp[warp_id] = val;
    }
    __syncthreads();
    float warp_val = tid < warp_nums ? warp[tid] : 0;
    return warpReduce<ReductionOp, T>(warp_val);
}